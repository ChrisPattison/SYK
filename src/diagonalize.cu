#include "hip/hip_runtime.h"
#include "diagonalize.hpp"

#include <cassert>
#include <algorithm>
#include <complex>
#include <exception>

#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


namespace syk {

GpuEigenValSolver::GpuEigenValSolver() {
    if(hipsolverDnCreate(&handle_)) { throw std::runtime_error("Failed to init cusolver"); }
}

GpuEigenValSolver::~GpuEigenValSolver() {
    hipsolverDnDestroy(handle_);
}


// std::vector<double> GpuEigenValSolver::eigenvals(const MatrixType& matrix) {
//     assert(matrix.cols() == matrix.rows());
//     int cols = matrix.cols();
//     thrust::device_vector<double> result(cols);

//     thrust::host_vector<hipDoubleComplex> host_A(matrix.size());
//     std::transform(matrix.data(), matrix.data() + matrix.size(), host_A.begin(), [](auto v) { return make_double2(v.real(), v.imag()); });
//     thrust::device_vector<hipDoubleComplex> A = host_A;
//     hipDeviceSynchronize();

//     // Work size query
//     int lwork;
//     auto status = hipsolverDnZheevd_bufferSize(handle_, 
//         HIPSOLVER_EIG_MODE_NOVECTOR, HIPBLAS_FILL_MODE_LOWER, 
//         cols, thrust::raw_pointer_cast(A.data()), cols, thrust::raw_pointer_cast(result.data()), &lwork);
//     hipDeviceSynchronize();
//     if(status) { throw std::runtime_error("Failed to get cusolver buffer size"); }

//     // Diagonalize
//     // TODO: Check storage order and if zgeev is expecting square matrix 
//     thrust::device_vector<hipDoubleComplex> work(lwork);
//     int device_info;
//     // status = hipsolverDnZheevd(handle_, HIPSOLVER_EIG_MODE_NOVECTOR, HIPBLAS_FILL_MODE_LOWER,
//     //     cols, thrust::raw_pointer_cast(A.data()), cols, 
//     //     thrust::raw_pointer_cast(result.data()), thrust::raw_pointer_cast(work.data()), work.size(), &device_info);
//     // hipDeviceSynchronize();
//     //if(device_info != 0 || status != 0) { throw std::runtime_error("Failed to run Zheevd"); }
   
//     std::vector<double> host_result(result.size());
//     thrust::copy(result.begin(), result.end(), host_result.begin());
//     hipDeviceSynchronize();
//     return host_result;
// }

std::vector<double> GpuEigenValSolver::eigenvals(const MatrixType& matrix) {
    assert(matrix.cols() == matrix.rows());
    int cols = matrix.cols();
    thrust::device_vector<float> result(cols);

    thrust::host_vector<hipFloatComplex> host_A(matrix.size());
    std::transform(matrix.data(), matrix.data() + matrix.size(), host_A.begin(), [](auto v) { return make_float2(static_cast<float>(v.real()), static_cast<float>(v.imag())); });
    thrust::device_vector<hipFloatComplex> A = host_A;
    hipDeviceSynchronize();

    // Work size query
    int lwork;
    auto status = hipsolverDnCheevd_bufferSize(handle_, 
        HIPSOLVER_EIG_MODE_NOVECTOR, HIPBLAS_FILL_MODE_LOWER, 
        cols, thrust::raw_pointer_cast(A.data()), cols, thrust::raw_pointer_cast(result.data()), &lwork);
    hipDeviceSynchronize();
    if(status) { throw std::runtime_error("Failed to get cusolver buffer size"); }

    // Diagonalize
    // TODO: Check storage order and if zgeev is expecting square matrix 
    thrust::device_vector<hipFloatComplex> work(lwork);
    thrust::device_vector<int> device_info(1);
    status = hipsolverDnCheevd(handle_, HIPSOLVER_EIG_MODE_NOVECTOR, HIPBLAS_FILL_MODE_LOWER,
        cols, thrust::raw_pointer_cast(A.data()), cols, 
        thrust::raw_pointer_cast(result.data()), 
        thrust::raw_pointer_cast(work.data()), lwork, 
        thrust::raw_pointer_cast(device_info.data()));
    hipDeviceSynchronize();
    if(device_info[0] != 0 || status != 0) { throw std::runtime_error("Failed to run Zheevd"); }
   
    thrust::host_vector<float> host_result = result;
    hipDeviceSynchronize();
    std::vector<double> stl_result(host_result.size());
    std::copy(host_result.begin(), host_result.end(), stl_result.begin());
    return stl_result;
}
}
