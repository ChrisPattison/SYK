#include "hip/hip_runtime.h"
#include "diagonalize.hpp"

#include <cassert>
#include <algorithm>
#include <complex>
#include <exception>

#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


namespace syk {

#if DIAG_SINGLE_PRECISION
    using float_t = float;
    using cuComplex_t = hipFloatComplex;
    #define PRECISION_HEEVD_bufferSize hipsolverDnCheevd_bufferSize
    #define PRECISION_HEEVD hipsolverDnCheevd
    #define PRECISION_MAKE_FLOAT2 make_float2
#else
    using float_t = double;
    using cuComplex_t = hipDoubleComplex;
    #define PRECISION_HEEVD_bufferSize hipsolverDnZheevd_bufferSize
    #define PRECISION_HEEVD hipsolverDnZheevd
    #define PRECISION_MAKE_FLOAT2 make_double2
#endif


GpuEigenValSolver::GpuEigenValSolver() {
    if(hipsolverDnCreate(&handle_)) { throw std::runtime_error("Failed to init cusolver"); }
}

GpuEigenValSolver::~GpuEigenValSolver() {
    hipsolverDnDestroy(handle_);
}


// std::vector<double> GpuEigenValSolver::eigenvals(const MatrixType& matrix) {
//     assert(matrix.cols() == matrix.rows());
//     int cols = matrix.cols();
//     thrust::device_vector<double> result(cols);

//     thrust::host_vector<hipDoubleComplex> host_A(matrix.size());
//     std::transform(matrix.data(), matrix.data() + matrix.size(), host_A.begin(), [](auto v) { return make_double2(v.real(), v.imag()); });
//     thrust::device_vector<hipDoubleComplex> A = host_A;
//     hipDeviceSynchronize();

//     // Work size query
//     int lwork;
//     auto status = hipsolverDnZheevd_bufferSize(handle_, 
//         HIPSOLVER_EIG_MODE_NOVECTOR, HIPBLAS_FILL_MODE_LOWER, 
//         cols, thrust::raw_pointer_cast(A.data()), cols, thrust::raw_pointer_cast(result.data()), &lwork);
//     hipDeviceSynchronize();
//     if(status) { throw std::runtime_error("Failed to get cusolver buffer size"); }

//     // Diagonalize
//     // TODO: Check storage order and if zgeev is expecting square matrix 
//     thrust::device_vector<hipDoubleComplex> work(lwork);
//     int device_info;
//     // status = hipsolverDnZheevd(handle_, HIPSOLVER_EIG_MODE_NOVECTOR, HIPBLAS_FILL_MODE_LOWER,
//     //     cols, thrust::raw_pointer_cast(A.data()), cols, 
//     //     thrust::raw_pointer_cast(result.data()), thrust::raw_pointer_cast(work.data()), work.size(), &device_info);
//     // hipDeviceSynchronize();
//     //if(device_info != 0 || status != 0) { throw std::runtime_error("Failed to run Zheevd"); }
   
//     std::vector<double> host_result(result.size());
//     thrust::copy(result.begin(), result.end(), host_result.begin());
//     hipDeviceSynchronize();
//     return host_result;
// }

std::vector<double> GpuEigenValSolver::eigenvals(const MatrixType& matrix) {
    assert(matrix.cols() == matrix.rows());
    int cols = matrix.cols();
    thrust::device_vector<float_t> result(cols);

    thrust::host_vector<cuComplex_t> host_A(matrix.size());
    std::transform(matrix.data(), matrix.data() + matrix.size(), host_A.begin(), [](auto v) { return PRECISION_MAKE_FLOAT2(static_cast<float_t>(v.real()), static_cast<float_t>(v.imag())); });
    thrust::device_vector<cuComplex_t> A = host_A;
    hipDeviceSynchronize();

    // Work size query
    int lwork;
    auto status = PRECISION_HEEVD_bufferSize(handle_, 
        HIPSOLVER_EIG_MODE_NOVECTOR, HIPBLAS_FILL_MODE_LOWER, 
        cols, thrust::raw_pointer_cast(A.data()), cols, thrust::raw_pointer_cast(result.data()), &lwork);
    hipDeviceSynchronize();
    if(status) { throw std::runtime_error("Failed to get cusolver buffer size"); }

    // Diagonalize
    // TODO: Check storage order and if zgeev is expecting square matrix 
    thrust::device_vector<cuComplex_t> work(lwork);
    thrust::device_vector<int> device_info(1);
    status = PRECISION_HEEVD(handle_, HIPSOLVER_EIG_MODE_NOVECTOR, HIPBLAS_FILL_MODE_LOWER,
        cols, thrust::raw_pointer_cast(A.data()), cols, 
        thrust::raw_pointer_cast(result.data()), 
        thrust::raw_pointer_cast(work.data()), lwork, 
        thrust::raw_pointer_cast(device_info.data()));
    hipDeviceSynchronize();
    if(device_info[0] != 0 || status != 0) { throw std::runtime_error("Failed to run Zheevd"); }
   
    thrust::host_vector<float_t> host_result = result;
    hipDeviceSynchronize();
    std::vector<double> stl_result(host_result.size());
    std::copy(host_result.begin(), host_result.end(), stl_result.begin());
    return stl_result;
}
}
